#include "hip/hip_runtime.h"
/* Copyright 2018 Stanford
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "taso/ops.h"
#include "taso/cuda_helper.h"
using namespace taso;

Model::Model()
: isTraining(false), print_cost(false)
{
  //int* a = (int*) malloc(sizeof(int) * 8);
  checkCUDA(hipSetDevice(0));
  checkCUDNN(hipdnnCreate(&dnn));
  checkCUDA(hipblasCreate(&blas));
  workSpaceSize = WORK_SPACE_SIZE;
  global_unique_id = 100;
  checkCUDA(hipMalloc(&workSpace, workSpaceSize));
  // printf("handle.workSpace = 0x%x\n", workSpace);
  // create all descriptors
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&biasTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&scaleTensor));
  checkCUDNN(hipdnnCreateFilterDescriptor(&filterDesc));
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convDesc));
  checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
  checkCUDNN(hipdnnCreateActivationDescriptor(&actiDesc));
  checkCUDNN(hipdnnCreateOpTensorDescriptor(&opDesc));
  // allocate tensors for measuring performance
  checkCUDA(hipMalloc(&inputPtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&biasPtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&outputPtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&filterPtr, MAX_TENSOR_SIZE));
  // create tensors for batch norm
  checkCUDA(hipMalloc(&scalePtr, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&runningMean, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&runningVar, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&saveMean, MAX_TENSOR_SIZE));
  checkCUDA(hipMalloc(&saveVar, MAX_TENSOR_SIZE));
  // create cuda events
  checkCUDA(hipEventCreate(&startEvent));
  checkCUDA(hipEventCreate(&endEvent));
}

float Model::measure_oplist_runtime(const std::vector<OpBase*>& opBaseList)
{
  // warmup
  for (int times = 0; times < 100; times++)
    for (int i = 0; i < opBaseList.size(); i++)
      opBaseList[i]->forward();

  // Estimate time per pass
  float time_taken;
  checkCUDA(hipDeviceSynchronize());
  checkCUDA(hipEventRecord(startEvent));
  for (int times = 0; times < 10; times++)
    for (int i = 0; i < opBaseList.size(); i++)
      opBaseList[i]->forward();
  checkCUDA(hipEventRecord(endEvent));
  checkCUDA(hipEventSynchronize(endEvent));
  hipEventElapsedTime(&time_taken, startEvent, endEvent);
  float time_per_pass = time_taken / 10.0;

  int num_minutes = 2;
  int num_passes = num_minutes * 60 * 1000 / time_per_pass;

  // Run GPU to stable temperature
  for (int times = 0; times < num_passes; times++)
    for (int i = 0; i < opBaseList.size(); i++)
      opBaseList[i]->forward();

  const int num_runs = 500;
  
  // measure runtime
  float total_time = 0;
  for (int times = 0; times < num_runs; times++) {
    float milliseconds;
    checkCUDA(hipDeviceSynchronize());
    checkCUDA(hipEventRecord(startEvent));
    for (int i = 0; i < opBaseList.size(); i++)
      opBaseList[i]->forward();
    checkCUDA(hipEventRecord(endEvent));
    checkCUDA(hipEventSynchronize(endEvent));
    hipEventElapsedTime(&milliseconds, startEvent, endEvent);
    total_time += milliseconds;
  }
  return total_time / num_runs;
}

void* Model::allocate_memory(size_t size, const DATATYPE* data_initial)
{
  void* ptr;
  checkCUDA(hipMalloc(&ptr, size));
  if (data_initial != NULL) {
    checkCUDA(hipMemcpy(ptr, data_initial, size, hipMemcpyDefault));
  }
  return ptr;
}

bool Model::copy_memory(DATATYPE* dst, const DATATYPE* src, size_t size)
{
  checkCUDA(hipMemcpy(dst, src, size, hipMemcpyDefault));
  return true;
}
